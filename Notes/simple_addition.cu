#include <stdio.h>
#include <hip/hip_runtime.h>


__global__
void vecAddKernel(float *A_d, float *B_d, float *C_d, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N) 
        C_d[i] = A_d[i] + B_d[i];
}


__host__
void vecAdd(float *A_d, float *B_d, float *C_d, int N)
{
    dim3 DimGrid(ceil(N/256.0), 1, 1);
    dim3 DimBlock(256, 1, 1);
    vecAddKernel<<<DimGrid, DimBlock>>>(A_d, B_d, C_d, N);
} 


int main()
{
    int N = 1024;
    int size = N * sizeof(float);

    // Allocate memory on the host
    float *A_h = (float*)malloc(size);
    float *B_h = (float*)malloc(size);
    float *C_h = (float*)malloc(size);

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        A_h[i] = i;
        B_h[i] = N - i;
    }

    // Allocate memory on the device
    float *A_d, *B_d, *C_d;
    hipMalloc((void**)&A_d, size);
    hipMalloc((void**)&B_d, size);
    hipMalloc((void**)&C_d, size);

    // Copy data from host to device
    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    // Call the vector addition function
    vecAdd(A_d, B_d, C_d, N);

    // Copy the result back to the host
    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    // Verify the result
    for (int i = 0; i < N; i++) {
        if (C_h[i] != A_h[i] + B_h[i]) {
            printf("Error at index %d: Expected %f, got %f\n", i, A_h[i] + B_h[i], C_h[i]);
            break;
        }
    }
    printf("All values are correct!\n");

    // Free memory
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    free(A_h);
    free(B_h);
    free(C_h);

    return 0;
}
